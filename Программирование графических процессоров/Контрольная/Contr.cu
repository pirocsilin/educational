#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
using namespace std;

float host_MulMatrixAndMatrix(int *, int *, int *);
int CheckRezult(int *, int *);
void InitMatrix(int *);
void PrintM(int *);

// Test data
// WIDHT: 500  800  1000  1200  1500  1800  2000  2500

#define WIDHT 2500      // Размер матрицы: WIDHT х WIDHT
#define THREADS 16      // Нитей на блок:  THREADS х THREADS
#define NUM_STREAMS 30  // Количество потоков

__global__ void MulMatrixAndMatrix(int *A, int *B, int *C, int Widht, int Height){

    int Tx = threadIdx.x;                                  // Номер столбца нити в блоке
    int Ty = threadIdx.y;                                  // Номер строки нити в блоке

    int Glob_y = (blockDim.y * blockIdx.y + Ty) * Widht;   // Индекс начала строки матрицы
    int Glob_x = blockDim.x * blockIdx.x + Tx;             // Индекс столбца матрицы

    int sum = 0;
    __shared__ int shared_A[THREADS][THREADS];
    __shared__ int shared_B[THREADS][THREADS];

    for(int i = 0; i < gridDim.x; i++){                 // Цикл по фрагментам матрицы

        int dy = i * blockDim.y * Widht;                // Смещение по столбцам и строкам
        int dx = i * blockDim.x;                        // в зависимости от фрагмента
        
        if(dx + Tx < Widht && Glob_y < Widht * Height)  // Если не выходим за границы
            shared_A[Ty][Tx] = A[Glob_y + dx + Tx];     // то пишем в разделяемую память

        if(Glob_x < Widht)
            shared_B[Ty][Tx] = B[Glob_x + dy + Ty * Widht];
        
        __syncthreads();                                // Ждем пока все нити загрузят данные

        if(Glob_x < Widht && Glob_y < Widht * Height)
            for(int k = 0; k < blockDim.x; k++){        // shared_A * shared_B
                int a = shared_A[Ty][k];
                int b = shared_B[k][Tx];
                sum += a * b;
            }

        __syncthreads();                                // Ждем пока все нити выполнят расчет
    }
    if(Glob_x < Widht && Glob_y < Widht * Height)       
        C[Glob_y + Glob_x] = sum;                       // Сохраняем результат
}

float MulMatrixOnMatrix(int *A, int *B, int *C, int *A_dev, int *B_dev, int *C_dev){
    
    hipStream_t *streams = new hipStream_t[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; i++){
        hipStreamCreate(&streams[i]);
    }

    int SIZE_CHUNK = (WIDHT / NUM_STREAMS) * WIDHT;
    int STEP = SIZE_CHUNK;

    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Копируем матрицы С и В на устройство целиком
    hipMemcpy(C_dev, C, WIDHT * WIDHT * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B, WIDHT * WIDHT * sizeof(int), hipMemcpyHostToDevice);

    for(int i = 0; i < NUM_STREAMS; i++){
        if(i == NUM_STREAMS - 1){
            SIZE_CHUNK = (WIDHT - WIDHT / NUM_STREAMS * (NUM_STREAMS - 1)) * WIDHT;
        }
        // Копируем часть матриц A на устройство
        hipMemcpyAsync(A_dev + i * STEP, A + i * STEP, sizeof(int) * SIZE_CHUNK,
                        hipMemcpyHostToDevice, streams[i] );

        dim3 Threads(THREADS, THREADS);
        // Рассчитываем Ширину и Высоту сетки блоков
        dim3 Blocks(WIDHT / THREADS + 1, SIZE_CHUNK / WIDHT / THREADS + 1);

        MulMatrixAndMatrix<<< Blocks, Threads, 0 , streams[i] >>>
                                (A_dev + i * STEP, 
                                B_dev, 
                                C_dev + i * STEP,
                                WIDHT, SIZE_CHUNK / WIDHT);

        // Копируем часть результата с девайса на хост
        hipMemcpyAsync(C + i * STEP, C_dev + i * STEP, sizeof(int) * SIZE_CHUNK,
                        hipMemcpyDeviceToHost, streams[i] );
    }
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete [] streams;

    return elapsedTime;
}

int main () {
    
    int *A, *B, *C;
    int *A_dev, *B_dev, *C_dev;

    hipHostMalloc((void**)&A, WIDHT * WIDHT * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&B, WIDHT * WIDHT * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&C, WIDHT * WIDHT * sizeof(int), hipHostMallocDefault);

    hipMalloc(&A_dev, WIDHT * WIDHT * sizeof(int));
    hipMalloc(&B_dev, WIDHT * WIDHT * sizeof(int));
    hipMalloc(&C_dev, WIDHT * WIDHT * sizeof(int));
    
    srand(1); InitMatrix(A);
    srand(2); InitMatrix(B);
    memset(C, 0, WIDHT * WIDHT * sizeof(int));

    float dev_Time = MulMatrixOnMatrix(A, B, C, A_dev, B_dev, C_dev);

    int *host_C = (int*)malloc(WIDHT * WIDHT * sizeof(int));

    float host_Time = host_MulMatrixAndMatrix(A, B, host_C);

    if(!CheckRezult(C, host_C)) 
        printf("\n ERROR CALCULATION !\n\n");
    else
        printf("\nMatrix size %d elem:\n\tDevice Time\t%.2f\n\tHost Time\t%.2f\n\tAcceleration\t%.2f\n\n", 
                   WIDHT * WIDHT, dev_Time, host_Time, host_Time / dev_Time);

    hipFree(A);    hipFree(A_dev);
    hipFree(B);    hipFree(B_dev);
    hipFree(C);    hipFree(C_dev);
    free(host_C);

	return 0;
}

int CheckRezult(int *C_dev, int *C_host){
    int check = 1;
    for(int i = 0; i < WIDHT * WIDHT; i++)
        if(C_dev[i] != C_host[i]){
            check = 0;
            break;
        }
    
    return check;
}

float host_MulMatrixAndMatrix(int *A, int *B, int *C){

    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for(int i = 0; i < WIDHT; i++)
        for(int j = 0; j < WIDHT; j++){
            int sum = 0;
            for(int k = 0; k < WIDHT; k++)
                sum += A[i * WIDHT + k] * B[k * WIDHT + j];
            C[i * WIDHT + j] = sum;
        }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

void InitMatrix(int *A){
    for(int i=0; i < WIDHT; i++)
        for(int j=0; j < WIDHT; j++)
            A[i * WIDHT +j] = rand() % 5 +1;
}