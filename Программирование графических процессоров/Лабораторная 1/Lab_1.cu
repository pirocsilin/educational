#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

// Тестовые размеры:
// 1000 x 500   1000 x 1000     1500 x 1000     2000 x 1000
// 2000 x 1500  2500 x 1500     2500x2000       3000 x 2000

int N = 3000,               // Height (количество строк)
    M = 2000;               // Widht  (количество столбцов)

enum mode{                  // Выбор режима умножения:
    MATRIX_MUL_VECTOR,      // 1. Матрица на вектор (вектор - столбец)
    VECTOR_MUL_MATRIX       // 2. Вектор на матрицу (вектор - строка)
} 
MODE = MATRIX_MUL_VECTOR;
//MODE = VECTOR_MUL_MATRIX;

void PrintM(int *);
void InitMatrix(int *);
void TranspMatrix(int *);
void PrintVector(int *, int);
int* InitVector(int, int NotNull = 1);

__global__ void MulMatrixAndVector(int *A, int *V, int *C, int Widht, int Height, int pitch){

    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Получаем глобальный номер нити

    if(i < Height){                                 // Если глобальный номер очередной нити
                                                    // не больше количества строк, то получаем
        int *Row_A = (int*)((char*)A + i * pitch);  // очередную строку матрицы из памяти и 

        int Sum  = 0;
        for(int k = 0; k < Widht; k++){             // каждый её элемент умножаем с соотв-м
            Sum += Row_A[k] * V[k];                 // элементом вектора V. Результаты суммируем
        }                                           // и записываем как элемент вектора С.
        C[i] = Sum;
    }
}

int main () {
    
    int *A = (int*)malloc(N * M * sizeof(int));

    int *V = NULL, *C = NULL;               // указатели на исходные данные и результат
    int *A_dev, *V_dev, *C_dev;             // указатели для записи адресов в памяти GPU
    size_t pitch;                           // фактическая ширина строки, in Bytes

    InitMatrix(A);                          // Инициализация матрицы.

    switch (MODE){                          
        case MATRIX_MUL_VECTOR:             // Если умножаем матрицу на вектор, то 
            V = InitVector(M);              // вектор будем рассматривать как один столбец
            C = InitVector(N, 0);           // матрицы, тогда его длина = ширине матрицы,
            break;                          // а результирующий вектор = высоте матрицы. 

        case VECTOR_MUL_MATRIX:             // При умножении вектора на матрицу, вектор 
            TranspMatrix(A);                // рассматривают как однострочную матрицу, 
            swap(N, M);                     // тогда его следует умножать на столбцы матрицы A.
            V = InitVector(M);              // Но в этом случа доступ к памяти будет не
            C = InitVector(N, 0);           // эффективным. Поэтому транспонируем матрицу
            break;                          // и работаем с вектором как как в первом случае
    }

    //PrintM(A);
    //PrintVector(V, M);
    //PrintVector(C, N);

    // Выделяем память на Девайсе для векторов и матрицы
    hipMalloc(&V_dev, M * sizeof(int));
    hipMalloc(&C_dev, N * sizeof(int));
    hipMallocPitch(&A_dev, &pitch, M * sizeof(int), N);

    // Создание и инициализация объектов событий
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Запись события start в этой точке выполнения программы
    hipEventRecord(start, 0);

    // Копируем данные с хоста на устройство
    hipMemcpy(V_dev, V, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C_dev, C, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy2D(A_dev, pitch, A, M * sizeof(int), M * sizeof(int), N, hipMemcpyHostToDevice);
    
    int Threads = 128;
    int Blocks = N / Threads + 1;
    
    MulMatrixAndVector<<< Blocks, Threads >>>(A_dev, V_dev, C_dev, M, N, pitch);
    
    hipDeviceSynchronize();                        // ожидаем завершение выполнения всех нитей
    
    hipMemcpy(C, C_dev, N * sizeof(int), hipMemcpyDeviceToHost);  // копируем результат на хост

    hipEventRecord(stop, 0);                       // запись события stop в точке программы
    hipEventSynchronize(stop);                     // ожидание завершения всех асинхронных команд 
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" \n elapsedTime = %f\n", elapsedTime);

    // Очищаем память на устройстве
    hipFree(V_dev);
    hipFree(C_dev);
    hipFree(A_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

	return 0;
}

void TranspMatrix(int *A){
    int *T = (int*)malloc(N * M * sizeof(int));
    for(int i=0; i < M; i++)
        for(int j=0; j < N; j++)
            T[i * N + j] = A[j * M + i];
    memcpy(A, T, sizeof(T));
}

void InitMatrix(int *A){
    for(int i=0; i < N; i++)
        for(int j=0; j < M; j++)
            A[i * M +j] = rand() % 9 +1;
}

int * InitVector(int CountElem, int NotNull){
    int *V = (int*)malloc(CountElem * sizeof(int));
    if(NotNull)
        for(int i=0; i < CountElem; i++) { V[i] = 2; }
    else
        memset(V, 0, CountElem * sizeof(int));
    return V;
}

void PrintVector(int *V, int n){
    cout << endl;
    for(int i = 0; i < n; i++) 
        cout << " " << V[i]; 
    cout << endl;
}

void PrintM(int *Matrix){
    for(int i=0; i < N; i++){
        for(int j=0; j < M; j++)
            cout << " " << Matrix[i * M +j];
        cout << endl;
    }
}