#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define THREAD_PER_BLOCK 128

using namespace std;

// Тестовые размеры:
// 1000 x 500   1000 x 1000     1500 x 1000     2000 x 1000
// 2000 x 1500  2500 x 1500     2500x2000       3000 x 2000

int N = 1000,               // Height (количество строк)
    M = 500;                // Widht  (количество столбцов)

enum mode{                  // Выбор режима умножения:
    MATRIX_MUL_VECTOR,      // 1. Матрица на вектор (вектор - столбец)
    VECTOR_MUL_MATRIX       // 2. Вектор на матрицу (вектор - строка)
} 
MODE = MATRIX_MUL_VECTOR;
//MODE = VECTOR_MUL_MATRIX;

void PrintM(int *);
void InitMatrix(int *);
void TranspMatrix(int *);
void PrintVector(int *, int);
int* InitVector(int, int NotNull = 1);

__shared__ int Part_v[THREAD_PER_BLOCK];                    // Части вектора
__shared__ int c[THREAD_PER_BLOCK];                         // Запись результатов 

__global__ void MulMatrixAndVector(int *A, int *V, int *C, int Widht, int Height, int pitch){

    int Global_i = threadIdx.x + blockIdx.x * blockDim.x;   // Получаем глобальный номер нити
    
    int *Row_A = (int*)((char*)A + Global_i * pitch);       // Получаем первый элемент соотв-й строки матрицы А

    c[threadIdx.x] = 0;                                     // Каждая нить блока инициализирует нулем "свой"
    __syncthreads();                                        // элемент результирующего вектора в разделяемой памяти                                       
        
        for(int k = 0; k < Widht / blockDim.x + 1; k++){    // Рзбиваем вектор V на k - частей
            
            int i = threadIdx.x + k * blockDim.x;           // Получаем номер элемента вектора,
            
            if(i < Widht){                                  // если он не выходит за границы вектора,
                Part_v[threadIdx.x] = V[i];                 // то загружаем его в разделяемую память
            }
            __syncthreads();                                // Ожидаем пока все нити блока загрузят "свой" элемент
                                                            
            if(Global_i < Height){                          // Если Global_i не больше высоты матрицы, то
                for(int j = 0; j < blockDim.x; j++){        // умножаем полученную часть вектора на часть матрицы  
                        c[threadIdx.x] += Part_v[j] * Row_A[k * blockDim.x + j];    
                }
            }
            __syncthreads();                                // Ожидаем пока все нити блока закончат вычисления 
        }                                                   // перед загрузкой следующей порции данных.

        if(Global_i < Height){                              // Проверка выхода номера нити за границы матрицы
            C[Global_i] = c[threadIdx.x];                   // Копируем результат в глобальную память                   
        }
        __syncthreads();                                    // Ждем пока все нити закончат копирование
}

int main () {
    
    int *A = (int*)malloc(N * M * sizeof(int));

    int *V = NULL, *C = NULL;               // указатели на исходные данные и результат
    int *A_dev, *V_dev, *C_dev;             // указатели для записи адресов в памяти GPU
    size_t pitch;                           // фактическая ширина строки, in Bytes

    InitMatrix(A);                          // Инициализация матрицы.

    switch (MODE){                          
        case MATRIX_MUL_VECTOR:             // Если умножаем матрицу на вектор, то 
            V = InitVector(M);              // вектор будем рассматривать как один столбец
            C = InitVector(N, 0);           // матрицы, тогда его длина = ширине матрицы,
            break;                          // а результирующий вектор = высоте матрицы. 

        case VECTOR_MUL_MATRIX:             // При умножении вектора на матрицу, вектор 
            TranspMatrix(A);                // рассматривают как однострочную матрицу, 
            swap(N, M);                     // тогда его следует умножать на столбцы матрицы A.
            V = InitVector(M);              // Но в этом случа доступ к памяти будет не
            C = InitVector(N, 0);           // эффективным. Поэтому транспонируем матрицу
            break;                          // и работаем с вектором как как в первом случае
    }

    //PrintM(A);
    //PrintVector(V, M);

    // Выделяем память на Девайсе для векторов и матрицы
    hipMalloc(&V_dev, M * sizeof(int));
    hipMalloc(&C_dev, N * sizeof(int));
    hipMallocPitch(&A_dev, &pitch, M * sizeof(int), N);

    // Создание и инициализация объектов событий
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Запись события start в этой точке выполнения программы
    hipEventRecord(start, 0);

    // Копируем данные с хоста на устройство
    hipMemcpy(V_dev, V, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C_dev, C, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy2D(A_dev, pitch, A, M * sizeof(int), M * sizeof(int), N, hipMemcpyHostToDevice);
    
    int Threads = THREAD_PER_BLOCK;
    int Blocks = N / Threads + 1;
    
    MulMatrixAndVector<<< Blocks, Threads >>>(A_dev, V_dev, C_dev, M, N, pitch);
    
    hipDeviceSynchronize();                        // ожидаем завершение выполнения всех нитей
    
    hipMemcpy(C, C_dev, N * sizeof(int), hipMemcpyDeviceToHost);  // копируем результат на хост
    
    hipEventRecord(stop, 0);                       // запись события stop в точке программы
    hipEventSynchronize(stop);                     // ожидание завершения всех асинхронных команд 
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" \n elapsedTime = %f\n", elapsedTime);

    // Очищаем память на устройстве
    hipFree(V_dev);
    hipFree(C_dev);
    hipFree(A_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //PrintVector(C, N);

	return 0;
}

void TranspMatrix(int *A){
    int *T = (int*)malloc(N * M * sizeof(int));
    for(int i=0; i < M; i++)
        for(int j=0; j < N; j++)
            T[i * N + j] = A[j * M + i];
    memcpy(A, T, sizeof(T));
}

void InitMatrix(int *A){
    for(int i=0; i < N; i++)
        for(int j=0; j < M; j++)
            A[i * M +j] = rand() % 5 + 1;
}

int * InitVector(int CountElem, int NotNull){
    int *V = (int*)malloc(CountElem * sizeof(int));
    if(NotNull)
        for(int i=0; i < CountElem; i++) { V[i] = 2; }
    else
        memset(V, 0, CountElem * sizeof(int));
    return V;
}

void PrintVector(int *V, int n){
    cout << endl;
    for(int i = 0; i < n; i++) 
        cout << " " << V[i]; 
    cout << endl;
}

void PrintM(int *Matrix){
    for(int i=0; i < N; i++){
        for(int j=0; j < M; j++)
            cout << " " << Matrix[i * M +j];
        cout << endl;
    }
}