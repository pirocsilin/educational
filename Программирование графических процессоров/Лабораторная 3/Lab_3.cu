#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>
using namespace std;

int N = 2500,                 // Height (количество строк)
    M = 2500;                 // Widht  (количество столбцов)

enum mode{
    MATRIX_MUL_VECTOR,
    VECTOR_MUL_MATRIX
} 
MODE = MATRIX_MUL_VECTOR;
//MODE = VECTOR_MUL_MATRIX;

void PrintM(int *);
void InitMatrix(int *);
void TranspMatrix(int *);
void PrintVector(int *, int);
int* InitVector(int, int NotNull = 1);

__global__ void MulMatrixAndVector(int *A, int *V, int *C, int Widht, int Height){

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i < Height){

        int *Row_A = (int*)((char*)A + i * Widht * sizeof(int));

        int Sum  = 0;
        for(int k = 0; k < Widht; k++){
            Sum += Row_A[k] * V[k];
        }
        C[i] = Sum;
    }
}

void TimeCalculation(int *A, int *V, int *C, int NUM_STREAMS, float &MinTime, int &NumStreams){

    int *A_dev, *V_dev, *C_dev;

    hipMalloc(&V_dev, M * sizeof(int));
    hipMalloc(&C_dev, N * sizeof(int));
    hipMalloc(&A_dev, N * M * sizeof(int));

    hipMemcpy(V_dev, V, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C_dev, C, N * sizeof(int), hipMemcpyHostToDevice);

    hipStream_t *streams = new hipStream_t[NUM_STREAMS];

    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    for(int i = 0; i < NUM_STREAMS; i++){
        hipStreamCreate(&streams[i]);
    }

    int SIZE_CHUNK = (N / NUM_STREAMS) * M;             // Количество элментов в части матрицы
    int STEP = SIZE_CHUNK;

    hipEventRecord(start, 0);                          // Фиксируем время начала расчетов
    for(int i = 0; i < NUM_STREAMS; i++){

        if(i == NUM_STREAMS - 1){
            SIZE_CHUNK = (N - N / NUM_STREAMS * (NUM_STREAMS - 1)) * M;
        }

        // Копируем часть матрицы A на устройство
        hipMemcpyAsync(A_dev + i * STEP,               // Указатель на часть матрицы на устройстве 
                        A  +    i * STEP,               // Указатель на часть матрицы на хосте
                        sizeof(int) * SIZE_CHUNK,       // Количество копируемых байт 
                        hipMemcpyHostToDevice,         // Копируем с хоста на устройство
                        streams[i] );                   // Номер очередного потока

        int Threads = 128;
        int Blocks = SIZE_CHUNK / M / Threads + 1;

        MulMatrixAndVector<<< Blocks, Threads, 0 , streams[i] >>>( 
                                A_dev + i * STEP,       // Указатель на часть матрицы на устройстве
                                V_dev,                  // Указатель на вектор V на устройстве
                                C_dev + i * STEP / M,   // Указатель на часть вектора C на устройстве
                                M, SIZE_CHUNK / M );    // Ширина и Высота части матрицы
        
        // Копируем часть вектора C на хост
        hipMemcpyAsync(C + i * STEP / M,               // Указатель на часть вектора С на хосте
                        C_dev + i * STEP / M,           // Указатель на часть C_dev на устройстве
                        sizeof(int) * SIZE_CHUNK / M,   // Количество копируемых байт
                        hipMemcpyDeviceToHost,         // Копируем с устройства на хост
                        streams[i] );                   // Номер очередного потока
    }
    hipDeviceSynchronize();                            // Ожидаем завершение выполнения всех нитей

    hipEventRecord(stop, 0);                           // Записываем время окончания расчетов
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    //printf( "%d: %0.3f\n", NUM_STREAMS, elapsedTime);

    if(elapsedTime < MinTime){
        MinTime = elapsedTime; 
        NumStreams = NUM_STREAMS;
    }

    hipFree(V_dev);
    hipFree(C_dev);
    hipFree(A_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    for(int i = 0; i < NUM_STREAMS; i++){
        hipStreamDestroy(streams[i]);
    }
    delete [] streams;

    memset(C, 0, N * sizeof(int));
}

int main () {
    
    int *A;                                 
    hipHostMalloc((void**)&A, N * M * sizeof(int), hipHostMallocDefault);

    int *V = NULL, *C = NULL;               // указатели на исходные данные и результат

    InitMatrix(A);                          // Инициализация матрицы.

    switch (MODE){                          
        case MATRIX_MUL_VECTOR:             // Если умножаем матрицу на вектор, то 
            V = InitVector(M);              // вектор будем рассматривать как один столбец
            C = InitVector(N, 0);           // матрицы, тогда его длина = ширине матрицы,
            break;                          // а результирующий вектор = высоте матрицы. 

        case VECTOR_MUL_MATRIX:             // При умножении вектора на матрицу, вектор 
            TranspMatrix(A);                // рассматривают как однострочную матрицу, 
            swap(N, M);                     // тогда его следует умножать на столбцы матрицы A.
            V = InitVector(M);              // Но в этом случа доступ к памяти будет не
            C = InitVector(N, 0);           // эффективным. Поэтому транспонируем матрицу
            break;                          // и работаем с вектором как как в первом случае
    }

    float MinTime = FLT_MAX; int NumStreams;
    for(int Streams = 2; Streams < 500; Streams++){
        TimeCalculation(A, V, C, Streams, MinTime, NumStreams);
    }

    printf("\n MinTime: %.3f, NumStreams: %d\n\n", MinTime, NumStreams);

    hipFree(A);
    hipFree(V);
    hipFree(C);

	return 0;
}

void TranspMatrix(int *A){
    int *T = (int*)malloc(N * M * sizeof(int));
    for(int i=0; i < M; i++)
        for(int j=0; j < N; j++)
            T[i * N + j] = A[j * M + i];
    memcpy(A, T, sizeof(T));
}

void InitMatrix(int *A){
    for(int i=0; i < N; i++)
        for(int j=0; j < M; j++)
            A[i * M +j] = rand() % 5 +1;
}

int * InitVector(int CountElem, int NotNull){
    int *Vector;
    hipHostMalloc((void**)&Vector, CountElem * sizeof(int), hipHostMallocDefault);
    if(NotNull)
        for(int i=0; i < CountElem; i++) { Vector[i] = 2; }
    else
        memset(Vector, 0, CountElem * sizeof(int));
    return Vector;
}

void PrintVector(int *V, int n){
    cout << endl;
    for(int i = 0; i < n; i++) 
        cout << " " << V[i]; 
    cout << endl;
}

void PrintM(int *Matrix){
    for(int i=0; i < N; i++){
        for(int j=0; j < M; j++)
            cout << " " << Matrix[i * M +j];
        cout << endl;
    }
}